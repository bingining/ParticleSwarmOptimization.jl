#include <hip/hip_runtime.h>
#include <assert.h>
#define CUDA(_s) assert((_s) == hipSuccess)
#define BLK 128
#define THR 128
#define KCALL(f,...) {f<<<BLK,THR>>>(__VA_ARGS__); CUDA(hipGetLastError()); }

__global__ void _update32(int n, float* px, float* pv, float* pbest, float* gbest,
    double w, double l, double h, double vmin, double vmax, double r1, double r2) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < n){
        pv[i] = pv[i] * w + r1 * (pbest[i] - px[i]) + r2 * (gbest[i] - px[i]);
        pv[i] = pv[i] < vmin ? vmin : pv[i] > vmax ? vmax : pv[i];
        px[i] += pv[i];
        px[i] = px[i] < l ? l : px[i] > h ? h : px[i];
    }
}

__global__ void _update64(int n, double* px, double* pv, double* pbest, double* gbest,
    double w, double l, double h, double vmin, double vmax, double r1, double r2) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < n){
        pv[i] = pv[i] * w + r1 * (pbest[i] - px[i]) + r2 * (gbest[i] - px[i]);
        pv[i] = pv[i] < vmin ? vmin : pv[i] > vmax ? vmax : pv[i];
        px[i] += pv[i];
        px[i] = px[i] < l ? l : px[i] > h ? h : px[i];
    }
}


extern "C" {
    void update64(int n, double* px, double* pv, double* pbest, double* gbest, double w, double l, double h, double vmin, double vmax, double r1, double r2) KCALL(_update64, n, px, pv, pbest, gbest, w, l, h, vmin, vmax, r1, r2);
    void update32(int n, float* px, float* pv, float* pbest, float* gbest, double w, double l, double h, double vmin, double vmax, double r1, double r2) KCALL(_update32, n, px, pv, pbest, gbest, w, l, h, vmin, vmax, r1, r2);    
}
